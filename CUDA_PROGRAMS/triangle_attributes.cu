#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(float2, barycentrics, attribute barycentrics, );

rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float2, varTexCoord, attribute TEXCOORD, );
rtDeclareVariable(float3, varShadingNormal, attribute SHADING_NORMAL, );

rtBuffer<float3> vertexBuffer;
rtBuffer<float2> uvBuffer;
rtBuffer<uint3> indicesBuffer;
rtBuffer<float3> normalBuffer;

//rtBuffer<char> procedure;

__device__ void test(char* bas) {

}

RT_PROGRAM void triangle_attributes() {
	const uint3 v_idx = indicesBuffer[rtGetPrimitiveIndex()];
	const float3 v0 = vertexBuffer[v_idx.x];
	const float3 v1 = vertexBuffer[v_idx.y];
	const float3 v2 = vertexBuffer[v_idx.z];
	const float3 n0 = normalBuffer[v_idx.x];
	const float3 n1 = normalBuffer[v_idx.y];
	const float3 n2 = normalBuffer[v_idx.z];
	varShadingNormal = optix::cross(v1 - v0, v2 - v0);

	const float2 t0 = uvBuffer[v_idx.x];
	const float2 t1 = uvBuffer[v_idx.y];
	const float2 t2 = uvBuffer[v_idx.z];

	barycentrics = rtGetTriangleBarycentrics();

	const float3 Ng = n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y);
	varGeoNormal = optix::normalize(Ng);

	const float3 temp = make_float3(t1*barycentrics.x + t2 * barycentrics.y + t0 * (1.0f - barycentrics.x - barycentrics.y));
	varTexCoord = make_float2(temp.x, temp.y);
}