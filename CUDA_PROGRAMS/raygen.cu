#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include "app_config.h"
#include "payloads.h"
#include "util.h"

using namespace optix;

rtBuffer<float4, 2>   result_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(Matrix3x3, view, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float, focus, , );

rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(int, in_medium, , );
rtDeclareVariable(float3, medium_color, , );
rtDeclareVariable(float, medium_density, , );
rtDeclareVariable(float3, light_position, , );
rtDeclareVariable(float, light_intensity, , );
rtDeclareVariable(float, ground_radius, , );
rtDeclareVariable(float, atmosphere_radius, , );
rtDeclareVariable(float, rayleigh, , );
rtDeclareVariable(float, mia, , );

RT_PROGRAM void raygen()
{
	PerRayData pyld;
	pyld.radiance = make_float3(0.0);
	pyld.main_light_position = light_position;
	pyld.main_light_intensity = light_intensity;
	
	const float2 pixel = make_float2(theLaunchIndex);
	const float2 fragment = pixel + make_float2(0.5f);
	const float2 screen = make_float2(theLaunchDim);
	const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

	float aspectRatio = screen.x / screen.y;

	float3 direction = view * normalize(make_float3(-ndc.x * aspectRatio, -ndc.y, focus));

	pyld.origin = origin;
	optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
	rtTrace(sysTopObject, ray, pyld);

	if(in_medium == 1 && (pyld.last_state == OTHER || pyld.last_state == ATMOSPHERE)){
        float3 as = atmospheric_scatter(light_position, light_intensity, medium_density, rayleigh, mia, make_float3(0, 0, 0),
            origin, pyld.medium_last_pos, ground_radius, atmosphere_radius);
        if(as.x!=as.x) as.x = 0.0;
        if(as.y!=as.y) as.y = 0.0;
        if(as.z!=as.z) as.z = 0.0;
        pyld.radiance = pyld.radiance + as;
	}

	result_buffer[launch_index] = make_float4(pyld.radiance , 1.0f);
}