#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtBuffer<float3> vertexBuffer;
rtBuffer<float2> uvBuffer;
rtBuffer<uint3> indicesBuffer;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float2, varTexCoord, attribute TEXCOORD, );

RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex)
{
  const uint3 indices = indicesBuffer[primitiveIndex];

  const float3 v0 = vertexBuffer[indices.x];
  const float3 v1 = vertexBuffer[indices.y];
  const float3 v2 = vertexBuffer[indices.z];

  const float2 uv0 = uvBuffer[indices.x];
  const float2 uv1 = uvBuffer[indices.y];
  const float2 uv2 = uvBuffer[indices.z];

  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
  {
    if (rtPotentialIntersection(t))
    {
		const float alpha = 1.0f - beta - gamma;
		varGeoNormal = n;
		varTexCoord = uv0 * alpha + uv1 * beta + uv2 * gamma;
		rtReportIntersection(0);
    }
  }
}
