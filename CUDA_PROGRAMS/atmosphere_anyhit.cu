#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include "payloads.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// The shadow ray program for all materials with no cutout opacity.
RT_PROGRAM void atmosphere_anyhit()
{
  //thePrd.radiance = make_float3(0.0, 0.0, 0.0);
  thePrd.visible = true;
  thePrd.distance = 0;
  thePrd.last_state = ATMOSPHERE;
  //rtTerminateRay();
}
