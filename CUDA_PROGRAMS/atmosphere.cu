#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "app_config.h"
#include "payloads.h"
#include "util.h"

rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float2, varTexCoord, attribute TEXCOORD, );
rtDeclareVariable(optix::float3, varShadingNormal, attribute SHADING_NORMAL, );

rtDeclareVariable(int, texture_id, , );
rtDeclareVariable(float, density, , );
rtDeclareVariable(float, ground_radius, , );
rtDeclareVariable(float, atmosphere_radius, , );
rtDeclareVariable(float, inv_altitude, , );
rtDeclareVariable(float, rayleigh, , );
rtDeclareVariable(float, mia, , );

rtBuffer<char> color_graph;

using namespace optix;

RT_PROGRAM void atmosphere()
{
	PerRayData pyld;
	pyld.visible = true;
	pyld.main_light_position = thePrd.main_light_position;
    pyld.main_light_intensity = thePrd.main_light_intensity;

	float3 origin = theRay.origin + theRay.direction * theIntersectionDistance;
	float3 direction = normalize(pyld.main_light_position - origin);

	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varShadingNormal));
	float3 ffnormal = faceforward(world_shade_normal, -theRay.direction, world_shade_normal);

	optix::Ray density_ray(origin, theRay.direction, 0, 0.1, RT_DEFAULT_MAX);

	rtTrace(sysTopObject, density_ray, pyld);

	float3 tex_color = make_float3(optix::rtTex2D<float4>(texture_id, varTexCoord.x, varTexCoord.y));

    if(pyld.last_state == NONE){
        thePrd.radiance = pyld.radiance;
    } else {
            float3 as = atmospheric_scatter(thePrd.main_light_position, thePrd.main_light_intensity, density, rayleigh, mia, make_float3(0, 0, 0), origin,
                pyld.medium_last_pos, ground_radius, atmosphere_radius);
            thePrd.radiance = pyld.radiance + as ;
    }

    thePrd.distance = theIntersectionDistance;
    thePrd.medium_last_pos = origin;
    thePrd.last_state = ATMOSPHERE;
}


