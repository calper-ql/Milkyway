#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>

rtBuffer<float3> vertexBuffer;
rtBuffer<float2> uvBuffer;
rtBuffer<uint3> indicesBuffer;

RT_PROGRAM void boundingbox_triangle_indexed(int primitiveIndex, float result[6])
{
  const uint3 indices = indicesBuffer[primitiveIndex];

  const float3 v0 = vertexBuffer[indices.x];
  const float3 v1 = vertexBuffer[indices.y];
  const float3 v2 = vertexBuffer[indices.z];

  const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

  optix::Aabb *aabb = (optix::Aabb *) result;

  if (0.0f < area && !isinf(area))
  {
    aabb->m_min = fminf(fminf(v0, v1), v2);
    aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
  }
  else
  {
    aabb->invalidate();
  }
}
