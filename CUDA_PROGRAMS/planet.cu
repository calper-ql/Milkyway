#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "app_config.h"
#include "payloads.h"
#include "util.h"

rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float2, varTexCoord, attribute TEXCOORD, );
rtDeclareVariable(optix::float3, varShadingNormal, attribute SHADING_NORMAL, );

rtDeclareVariable(int, texture_id, , );

rtBuffer<char> color_graph;

using namespace optix;

RT_PROGRAM void planet()
{
	PerRayData pyld;
	pyld.visible = true;
	pyld.main_light_position = thePrd.main_light_position;
    pyld.main_light_intensity = thePrd.main_light_intensity;

	float3 origin = theRay.origin + theRay.direction * theIntersectionDistance;
	float3 direction = normalize(origin-thePrd.main_light_position);

	//float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	//float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varShadingNormal));
	//float3 ffnormal = faceforward(world_shade_normal, -theRay.direction, world_shade_normal);

    float shadow = calculate_sphere_shadow(make_float3(0.0), origin, thePrd.main_light_position, 0.04, 0.00);

	float Ldist = length(thePrd.main_light_position - origin);
	optix::Ray shadow_ray(origin, direction, 1, 1.00, Ldist);
	rtTrace(sysTopObject, shadow_ray, pyld);


	if (pyld.visible) {
		thePrd.radiance = make_float3(optix::rtTex2D<float4>(texture_id, varTexCoord.x, varTexCoord.y)) * shadow;
	} else {
		thePrd.radiance = make_float3(optix::rtTex2D<float4>(texture_id, varTexCoord.x, varTexCoord.y)) * shadow * 0.8;
	}

    thePrd.distance = theIntersectionDistance;
    thePrd.medium_last_pos = origin;
    thePrd.last_state = OTHER;
}


