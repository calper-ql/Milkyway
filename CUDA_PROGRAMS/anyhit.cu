#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include "payloads.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// The shadow ray program for all materials with no cutout opacity.
RT_PROGRAM void anyhit()
{
  thePrd.radiance = make_float3(0.0, 0.0, 0.0);
  thePrd.visible = false;
  thePrd.distance = 0;
  thePrd.last_state = OTHER;
  rtTerminateRay();
}
